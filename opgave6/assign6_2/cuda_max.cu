#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <float.h>
#include <iostream>

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

__device__ int NearestPowerOf2(int n)
{
  if (!n) return n;  //(0 == 2^0)
 
  int x = 1;
  while(x < n)
    {
      x <<= 1;
    }
  return x;
}


// standard binary tree reduction cude method
__global__ void maxKernel() {
  int  thread2;
  float temp;
  __shared__ float max[BLOCK_SIZE];
   
  int nTotalThreads = NearestPowerOf2(blockDim.x);	// Total number of threads, rounded up to the next power of two
   
  while(nTotalThreads > 1)
  {
    int halfPoint = (nTotalThreads >> 1);	// divide by two
    // only the first half of the threads will be active.
   
    if (threadIdx.x < halfPoint)
    {
     thread2 = threadIdx.x + halfPoint;
   
     // Skipping the fictious threads blockDim.x ... blockDim_2-1
     if (thread2 < blockDim.x)
       {
   
        temp = max[thread2];
        if (temp > max[threadIdx.x]) 
           max[threadIdx.x] = temp;
       }
    }
    __syncthreads();
   
    // Reducing the binary tree size by two:
    nTotalThreads = halfPoint;
  }
}

// TODO create working compute function that returns the max value of an array
float *computeMaxCuda(int length){
    
  float* d_list = NULL;
  float* d_max = NULL;
  float* maxVal = NULL;
  int tpb = 128;
  float list[length];
  
  srand(time(NULL));
  //TODO make this run in parallel
  for(int i = 0; i< length); i++)
    list[i] = (float)rand()/((float)RAND_MAX/FLT_MAX);
  
	// Alloc space on the device.
	// Is this the right amount?
	checkCudaCall(hipMalloc((void **) &d_list, length * sizeof(float)));
	checkCudaCall(hipMalloc((void **) &d_max, sizeof(float)));
	
	// TODO make the right call
  maxKernel<<<(int) ceil((double) length / (double) tpb), tpb>>>();

  // copy resulting max back to main memory
  checkCudaCall(hipMemcpy(maxVal, d_max, sizeof(float), hipMemcpyDeviceToHost));

	// Free device mem.
	checkCudaCall(hipFree(d_list));
	checkCudaCall(hipFree(d_max));
	
	return maxVal;
}
