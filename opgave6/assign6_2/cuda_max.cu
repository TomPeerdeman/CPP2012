#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <float.h>
#include <iostream>

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


// TODO create some function that adds values.
__global__ void maxKernel() {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
	
}

// TODO create compute function that returns the max value of an array
float *computeMaxCuda(int length){
  srand(time(NULL));
  float list[length];
  //TODO make this run in parallel
  for(int i = 0; i< length); i++)
    list[i] = (float)rand()/((float)RAND_MAX/FLT_MAX);
    
    
	// Alloc space on the device. TODO alloc right amount
	checkCudaCall(hipMalloc((void **) &dOld, i_max * sizeof(float)));
	
	// Copy from main mem to device mem. TODO alloc right amount
	checkCudaCall(hipMemcpy(dOld, hOld, i_max*sizeof(float), hipMemcpyHostToDevice));
	checkCudaCall(hipMemcpy(dCur, hCur, i_max*sizeof(float), hipMemcpyHostToDevice));
	
	// TODO make the right call
  maxKernel<<<(int) ceil((double) i_max / (double) tpb), tpb>>>(i_max, dOld, dCur, dNext);
	
	// Copy back the result from device mem to main mem. TODO copy right amount back
	checkCudaCall(hipMemcpy(hCur, dCur, i_max * sizeof(float), hipMemcpyDeviceToHost));
	
	// Free device mem.
	checkCudaCall(hipFree(dOld));
	
	return maxVal;
}
